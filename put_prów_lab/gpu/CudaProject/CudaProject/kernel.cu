#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <cmath>
#include <chrono>
#include <iomanip>

__global__ void computeOutput(float* input, float* output, int N, int R, int k) {
    int i = blockIdx.y * blockDim.y + threadIdx.y + R;
    int j = blockIdx.x * blockDim.x + threadIdx.x + R;

    if (i < N - R && j < N - R) {
        for (int idx = 0; idx < k; ++idx) {
            float sum = 0.0f;
            for (int di = -R; di <= R; ++di) {
                for (int dj = -R; dj <= R; ++dj) {
                    sum += input[(i + di) * N + (j + dj)];
                }
            }
            output[(i - R) * (N - 2 * R) + (j - R) + idx * (N - 2 * R) * (N - 2 * R)] = sum;
        }
    }
}

void computeSequential(float* input, float* output, int N, int R) {
    for (int i = R; i < N - R; ++i) {
        for (int j = R; j < N - R; ++j) {
            float sum = 0.0f;
            for (int di = -R; di <= R; ++di) {
                for (int dj = -R; dj <= R; ++dj) {
                    sum += input[(i + di) * N + (j + dj)];
                }
            }
            output[(i - R) * (N - 2 * R) + (j - R)] = sum;
        }
    }
}

int main() {
    // Parametry problemu
    int R = 1;    // Promień R
    int k = 1;    // Liczba wyników obliczanych przez jeden wątek

    for (int N = 640; N <= 1536; N += 128) {
        for (int multi = 2; multi <= 16; multi *= 2) {
            R = multi;
            k = multi;
            std::cout << "R=" << R << ", k=" << k << "\n";
            float* inputHost = new float[N * N];
            float* outputHostCPU = new float[(N - 2 * R) * (N - 2 * R)];
            float* outputHostGPU = new float[(N - 2 * R) * (N - 2 * R) * k];

            // Inicjalizacja danych wejściowych na CPU
            for (int i = 0; i < N * N; ++i) {
                inputHost[i] = static_cast<float>(rand()) / RAND_MAX;
            }

            // Obliczenia sekwencyjne dla porównania
            auto startCPU = std::chrono::high_resolution_clock::now();
            computeSequential(inputHost, outputHostCPU, N, R);
            auto endCPU = std::chrono::high_resolution_clock::now();
            std::chrono::duration<float> durationCPU = endCPU - startCPU;
            std::cout << "CPU Time for N=" << N << ": " << durationCPU.count() << "s\n";

            // Przygotowanie wskaźników na dane na GPU
            float* inputDevice, * outputDevice;
            hipMalloc((void**)&inputDevice, N * N * sizeof(float));
            hipMalloc((void**)&outputDevice, (N - 2 * R) * (N - 2 * R) * k * sizeof(float));

            // Kopiowanie danych z CPU do GPU
            hipMemcpy(inputDevice, inputHost, N * N * sizeof(float), hipMemcpyHostToDevice);

            // Określenie rozmiaru siatki wątków i bloków

            dim3 blockSize(32, 32);
            dim3 gridSize((N - 2 * R + blockSize.x - 1) / blockSize.x, (N - 2 * R + blockSize.y - 1) / blockSize.y);

            // Wywołanie kernela
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start, 0);

            computeOutput << <gridSize, blockSize >> > (inputDevice, outputDevice, N, R, k);

            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);

            float elapsedTime;
            hipEventElapsedTime(&elapsedTime, start, stop);
            std::cout << "GPU Time for N=" << N << ": " << elapsedTime / 1000.0f << "s\n";

            // Kopiowanie wyników z GPU do CPU
            hipMemcpy(outputHostGPU, outputDevice, (N - 2 * R) * (N - 2 * R) * k * sizeof(float), hipMemcpyDeviceToHost);

            // Obliczanie FLOPS
            int numOps = (N - 2 * R) * (N - 2 * R) * (2 * R + 1) * (2 * R + 1);
            float flops = numOps / (elapsedTime / 1000.0f);
            printf("FLOP/s for N=%d: %.2e\n", N, flops);

            // Sprawdzenie poprawności obliczeń GPU poprzez porównanie z wynikami CPU
            float maxError = 0.0f;
            for (int i = 0; i < (N - 2 * R) * (N - 2 * R); ++i) {
                maxError = fmax(maxError, fabs(outputHostCPU[i] - outputHostGPU[i]));
            }

            if (maxError < 1e-5) {
                std::cout << "Poprawnosc obliczen GPU zostala zweryfikowana." << std::endl;
            }
            else {
                std::cout << "Blad obliczeń GPU! Maksymalny błąd: " << maxError << std::endl;
            }

            // Zwolnienie pamięci na GPU
            hipFree(inputDevice);
            hipFree(outputDevice);

            // Zwolnienie pamięci na CPU
            delete[] inputHost;
            delete[] outputHostCPU;
            delete[] outputHostGPU;

            std::cout << "--------------------------------------------" << std::endl;
        }
       
    }

    return 0;
}

